#include "hip/hip_runtime.h"
#include "b.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

void PTWDSP_check_memory_usage()
{
	size_t free;
	size_t total;
	hipMemGetInfo(&free, &total);
	size_t used = total - free;
	std::cout << "Xavier momory usage:used = " << used / (1024 * 1024) << " MB, "
			  << "free = " << free / (1024 * 1024) << " MB, "
			  << "total = " << total / (1024 * 1024) << " MB " << std::endl;
}

void read_raw_file_16_int(char *filename, unsigned short *bitmap_data, long *width, long *heigth, int header)
{
	FILE *fp_raw;
	long x, y;
	char *ptr;
	//header表示有無標頭
	fp_raw = fopen(filename, "rb");

	if (fp_raw == NULL)
	{
		exit(-1);
	}

	ptr = (char *)bitmap_data;
	if (header == 1)
		for (int i = 0; i < 16; i++)
			fgetc(fp_raw);

	for (y = 0; y < *heigth; y++)
	{
		for (x = 0; x < *width * 2; x++)
		{
			ptr[y * (*width) * 2 + x] = fgetc(fp_raw);
		}
	}
	fclose(fp_raw);
}

void write_raw_file(char *filename, unsigned short *bitmap_data, long *width, long *heigth)
{
	FILE *fp_raw;
	long x, y;
	char *ptr;
	char hdr_str[20];
	fp_raw = fopen(filename, "wb");
	if (fp_raw == NULL)
	{
		exit(-1);
	}

	memset((char *)hdr_str, 0, 20);
	hdr_str[8] = *width % 256;
	hdr_str[9] = (int)*width / 256;
	hdr_str[12] = *heigth % 256;
	hdr_str[13] = (int)*heigth / 256;
	fwrite(hdr_str, 1, 16, fp_raw);

	ptr = (char *)bitmap_data;

	for (y = 0; y < *heigth; y++)
	{
		for (x = 0; x < *width * 2; x++)
		{
			fputc(ptr[y * (*width) * 2 + x] >> 4, fp_raw);
		}
	}
	fclose(fp_raw);
}

__global__ void CalcImageConvolution(int mode, unsigned short *h_Result, unsigned short *h_Data, int *h_Kernel, long dataH, long dataW, long kernelH, long kernelW, long kernelY, long kernelX)
{
	int x, y, kx, ky, dx, dy;
	double sum;
	float normalize_factor;

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < dataH * dataW)
	{
		sum = 0.0;
		x = i % dataW;
		y = (int)(i / dataW);

		normalize_factor = (float)(kernelH * kernelW);

		for (ky = -(kernelH - kernelY - 1); ky <= kernelY; ky++)
		{
			for (kx = -(kernelW - kernelX - 1); kx <= kernelX; kx++)
			{
				dy = y + ky;
				dx = x + kx;
				if (dy < 0)
					sum += 0;
				else if (dx < 0)
					sum += 0;
				else if (dy >= dataH)
					sum += 0;
				else if (dx >= dataW)
					sum += 0;
				else
					sum += ((double)h_Data[dy * dataW + dx]) * ((double)h_Kernel[(kernelY + ky) * kernelW + (kernelX + kx)]);
			}
		}

		if (mode == 1)
			h_Result[y * dataW + x] = (unsigned short)(sum);
		else
			h_Result[y * dataW + x] = (unsigned short)((double)(sum / normalize_factor));
	}
}

void CalcImageConvolution_cuda(unsigned short *data, long w, long h, unsigned int frame_num)
{
	long kernelH, kernelW, kernelY, kernelX;
	unsigned short *kernel, *result;
	unsigned short *data_g, *result_g;
	char fname[FILENAME_MAX];
	int *kernel_g;
	kernelH = 3;
	kernelW = 3;
	kernelX = 1;
	kernelY = 1;

	//data = (unsigned short *)malloc(w*h * sizeof(unsigned short));
	kernel = (unsigned short *)malloc(w * h * sizeof(unsigned short));
	result = (unsigned short *)malloc(w * h * sizeof(unsigned short));
	PTWDSP_check_memory_usage();
	/*
	unsigned short kernel_3X3[] = { 1, 1, 1, 
					1, 1, 1,
					1, 1, 1};
	*/
	int kernel_3X3[] = {1, 0, -1,
						2, 0, -2,
						1, 0, -1};
	hipMalloc((void **)&data_g, w * h * sizeof(unsigned short));
	hipMalloc((void **)&kernel_g, kernelH * kernelW * sizeof(int));
	hipMalloc((void **)&result_g, w * h * sizeof(unsigned short));
	// Invoke kernel
	int threads_no = 512;
	int threadsPerBlock = threads_no;
	int blocksPerGrid = (h * w + threadsPerBlock - 1) / threadsPerBlock;
	hipMemcpy(data_g, data, h * w * sizeof(unsigned short), hipMemcpyHostToDevice);
	hipMemcpy(kernel_g, kernel_3X3, 3 * 3 * sizeof(int), hipMemcpyHostToDevice);

	sprintf(fname, "./Pic/NTUST_Xavier_test_%03u.raw", (unsigned)frame_num);
	write_raw_file((char *)fname, (unsigned short *)data, (long *)&w, (long *)&h);
	//write_raw_file((char *)"reverse.raw", (unsigned short *)data2, (long *)&w, (long *)&h);
	CalcImageConvolution<<<blocksPerGrid, threadsPerBlock>>>(1, result_g, data_g, kernel_g, h, w, kernelH, kernelW, kernelY, kernelX);

	hipMemcpy(result, result_g, h * w * sizeof(unsigned short), hipMemcpyDeviceToHost);
	//write_raw_file((char *)"NTUST_Xavier_test_con.raw", (unsigned short *)result, (long *)&w, (long *)&h);
	hipFree(data_g);
	hipFree(kernel_g);
	hipFree(result_g);
	//free(data);
	free(kernel);
	free(result);
}
